
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
#define numk 128
#define numv 1024

__global__ void cal(double *prob);

__device__ hipDoubleComplex point(double kappa, double V_0);

__device__ hipDoubleComplex cuConjugate(hipDoubleComplex x) {
  return make_hipDoubleComplex(hipCreal(x), -hipCimag((x)));
}

__device__ hipDoubleComplex cuConjugate(double x) {
  return make_hipDoubleComplex(x, 0);
}

__global__ void test(float kappa, float V_0) {
  auto pt = point(kappa, V_0);
  printf("here\n");
  printf("%f\t%f\n", hipCreal(pt), hipCimag(pt));
}

int main() {
  // const int numk = 100;
  // const int numv = 1000;
  double *d_prob;
  double *prob = (double *)malloc(numk * numv * sizeof(double));
  hipMalloc((void **)&d_prob, numk * numv * sizeof(double));
  cal<<<128, 256>>>(d_prob);
  hipDeviceSynchronize();
  auto err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error: %s\n", hipGetErrorString(err));
    exit(err);
  }
  hipMemcpy(prob, d_prob, numk * numv * sizeof(double),
             hipMemcpyDeviceToHost);
  // test<<<1, 1>>>(0.2341, 170);
  // cudaDeviceSynchronize();
  char res[7500001];
  int len = 0;
  for (int j = 0; j < numv; j++) {
    for (int i = 0; i < numk; i++) {
      len += sprintf(res + len, "%f\t%f\n", 0.001 + 1. / 128. * i,
                     prob[j * numk + i]);
    }
    len += sprintf(res + len, "\n\n");
  }
  FILE *fptr = fopen("prob.dat", "w");
  fprintf(fptr, "%s", res);
  fclose(fptr);
  return 0;
}

__global__ void cal(double *prob) {
  // printf("here\n");
  int i = blockIdx.x;
  for (int offset = 0; offset < 4; offset++) {
    int j = threadIdx.x * 4 + offset;
    if (i >= numk || j >= numv) {
      return;
    }
    double kappa = 0.001 + (1. / 128.) * i;
    double V_0 = 80 + 0.1 * j;
    auto pt = point(kappa, V_0);
    // printf("%f %f\n", pt.x, pt.y);
    // printf("%f\n", x);
    // printf("%f %f\n", kappa, V_0);
    prob[j * gridDim.x + i] = pt.x;
  }
  return;
}

__device__ hipDoubleComplex operator*(const hipDoubleComplex &z,
                                     const hipDoubleComplex &r) {
  return hipCmul(z, r);
}

__device__ hipDoubleComplex operator*(const hipDoubleComplex &z,
                                     const double &r) {
  return make_hipDoubleComplex(hipCreal(z) * r, hipCimag(z) * r);
}

__device__ hipDoubleComplex operator*(const double &r,
                                     const hipDoubleComplex &z) {
  return z * r;
}

__device__ hipDoubleComplex operator+(const hipDoubleComplex &z,
                                     const hipDoubleComplex &r) {
  return hipCadd(z, r);
}

__device__ hipDoubleComplex operator+(const hipDoubleComplex &z,
                                     const double &r) {
  return hipCadd(z, make_hipDoubleComplex(r, 0));
}

__device__ hipDoubleComplex operator+(const double &r,
                                     const hipDoubleComplex &z) {
  return z + r;
}

__device__ hipDoubleComplex operator-(const hipDoubleComplex &lhs, const hipDoubleComplex &rhs) {
    return make_hipDoubleComplex(lhs.x - rhs.x, lhs.y - rhs.y);
}

__device__ hipDoubleComplex operator/(const hipDoubleComplex &z,
                                     const double &a) {
  return z * (1 / a);
}
__device__ hipDoubleComplex operator/(const double &a,
                                     const hipDoubleComplex &z) {
  return a * cuConjugate(z) /
         (hipCreal(z) * hipCreal(z) - hipCimag(z) * hipCimag(z));
}

__device__ hipDoubleComplex operator/(const hipDoubleComplex &z,
                                     const hipDoubleComplex &a) {
  return z * cuConjugate(a) /
         (hipCreal(a) * hipCreal(a) - hipCimag(a) * hipCimag(a));
}

__device__ hipDoubleComplex exp(const hipDoubleComplex z) {
  double a = hipCreal(z);
  double b = hipCimag(z);
  return exp(a) * make_hipDoubleComplex(cos(b), sin(b));
}

__device__ hipDoubleComplex point(double kappa, double V_0) {
  // printf("here\n");
return -4.0960000000000009e-13*(kappa)/exp(make_hipDoubleComplex(0.0,2.0000000000000000e-02)*pow(2.0,(1.0/2.0))*pow(kappa*V_0,(1.0/2.0)))*exp(-2.0000000000000000e-02*pow(2.0,(1.0/2.0))*pow( V_0-kappa*V_0,(1.0/2.0)))*( (kappa)*(V_0)-(V_0))*kappa*exp(-2.0000000000000000e-02*(pow( V_0-kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))/( 2.4000000000000003e-07*(kappa)*( (kappa)*(V_0)-(V_0))*(V_0)+1.6000000000000000e-07*(kappa)*( (kappa)*(V_0)-(V_0))*exp(-2.0000000000000000e-02*(pow( V_0-kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*(V_0)+8.0000000000000002e-08*(kappa)*( (kappa)*(V_0)-(V_0))*exp(make_hipDoubleComplex(0.0,-2.0000000000000000e-02)*(pow(kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*(V_0)+4.0000000000000001e-08*pow((kappa),2.0)*exp(-4.0000000000000001e-02*(pow( V_0-kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*pow((V_0),2.0)+-1.6000000000000000e-07*(kappa)*( (kappa)*(V_0)-(V_0))*exp(-2.0000000000000000e-02*(pow( V_0-kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*exp(make_hipDoubleComplex(0.0,-2.0000000000000000e-02)*(pow(kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*(V_0)+8.0000000000000002e-08*pow((kappa),2.0)*exp(-2.0000000000000000e-02*(pow( V_0-kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*exp(make_hipDoubleComplex(0.0,-2.0000000000000000e-02)*(pow(kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*pow((V_0),2.0)+-8.0000000000000002e-08*pow((kappa),2.0)*exp(-2.0000000000000000e-02*(pow( V_0-kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*pow((V_0),2.0)+-4.0000000000000001e-08*pow((kappa),2.0)*exp(make_hipDoubleComplex(0.0,-2.0000000000000000e-02)*(pow(kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*pow((V_0),2.0)+-4.0000000000000001e-08*exp(-4.0000000000000001e-02*(pow( V_0-kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*pow( (kappa)*(V_0)-(V_0),2.0)*exp(make_hipDoubleComplex(0.0,-2.0000000000000000e-02)*(pow(kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))+make_hipDoubleComplex(0.0,-1.6000000000000000e-07)*(pow(kappa*V_0,(3.0/2.0)))*(pow( V_0-kappa*V_0,(1.0/2.0)))+make_hipDoubleComplex(0.0,-1.6000000000000000e-07)*(pow(kappa*V_0,(1.0/2.0)))*exp(-4.0000000000000001e-02*(pow( V_0-kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*(pow( V_0-kappa*V_0,(3.0/2.0)))+-4.0000000000000001e-08*pow( (kappa)*(V_0)-(V_0),2.0)*exp(make_hipDoubleComplex(0.0,-2.0000000000000000e-02)*(pow(kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))+4.0000000000000001e-08*exp(-4.0000000000000001e-02*(pow( V_0-kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*pow( (kappa)*(V_0)-(V_0),2.0)+8.0000000000000002e-08*(kappa)*exp(-4.0000000000000001e-02*(pow( V_0-kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*( (kappa)*(V_0)-(V_0))*exp(make_hipDoubleComplex(0.0,-2.0000000000000000e-02)*(pow(kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*(V_0)+8.0000000000000002e-08*pow( (kappa)*(V_0)-(V_0),2.0)*exp(-2.0000000000000000e-02*(pow( V_0-kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*exp(make_hipDoubleComplex(0.0,-2.0000000000000000e-02)*(pow(kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))+4.0000000000000001e-08*pow( (kappa)*(V_0)-(V_0),2.0)+-4.0000000000000001e-08*pow((kappa),2.0)*exp(-4.0000000000000001e-02*(pow( V_0-kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*exp(make_hipDoubleComplex(0.0,-2.0000000000000000e-02)*(pow(kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*pow((V_0),2.0)+2.4000000000000003e-07*(kappa)*exp(-4.0000000000000001e-02*(pow( V_0-kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*( (kappa)*(V_0)-(V_0))*(V_0)+4.0000000000000001e-08*pow((kappa),2.0)*pow((V_0),2.0)+make_hipDoubleComplex(0.0,1.6000000000000000e-07)*(pow(kappa*V_0,(1.0/2.0)))*(pow( V_0-kappa*V_0,(3.0/2.0)))+-8.0000000000000002e-08*pow( (kappa)*(V_0)-(V_0),2.0)*exp(-2.0000000000000000e-02*(pow( V_0-kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))+make_hipDoubleComplex(0.0,1.6000000000000000e-07)*(pow(kappa*V_0,(3.0/2.0)))*exp(-4.0000000000000001e-02*(pow( V_0-kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*(pow( V_0-kappa*V_0,(1.0/2.0))))*V_0/exp(make_hipDoubleComplex(0.0,-2.0000000000000000e-02)*(pow(kappa*V_0,(1.0/2.0)))*pow(2.0,(1.0/2.0)))*(V_0)/( make_hipDoubleComplex(0.0,-1.6000000000000000e-07)*pow( V_0-kappa*V_0,(3.0/2.0))*pow(kappa*V_0,(1.0/2.0))+-2.4000000000000003e-07*exp(-4.0000000000000001e-02*pow(2.0,(1.0/2.0))*pow( V_0-kappa*V_0,(1.0/2.0)))*kappa*V_0*( V_0-kappa*V_0)+-8.0000000000000002e-08*exp(make_hipDoubleComplex(0.0,2.0000000000000000e-02)*pow(2.0,(1.0/2.0))*pow(kappa*V_0,(1.0/2.0)))*kappa*V_0*( V_0-kappa*V_0)+4.0000000000000001e-08*exp(-4.0000000000000001e-02*pow(2.0,(1.0/2.0))*pow( V_0-kappa*V_0,(1.0/2.0)))*pow( V_0-kappa*V_0,2.0)+make_hipDoubleComplex(0.0,1.6000000000000000e-07)*exp(-4.0000000000000001e-02*pow(2.0,(1.0/2.0))*pow( V_0-kappa*V_0,(1.0/2.0)))*pow( V_0-kappa*V_0,(3.0/2.0))*pow(kappa*V_0,(1.0/2.0))+make_hipDoubleComplex(0.0,1.6000000000000000e-07)*pow( V_0-kappa*V_0,(1.0/2.0))*pow(kappa*V_0,(3.0/2.0))+4.0000000000000001e-08*exp(-4.0000000000000001e-02*pow(2.0,(1.0/2.0))*pow( V_0-kappa*V_0,(1.0/2.0)))*(kappa*kappa)*(V_0*V_0)+-4.0000000000000001e-08*exp(make_hipDoubleComplex(0.0,2.0000000000000000e-02)*pow(2.0,(1.0/2.0))*pow(kappa*V_0,(1.0/2.0)))*pow( V_0-kappa*V_0,2.0)+-2.4000000000000003e-07*kappa*V_0*( V_0-kappa*V_0)+1.6000000000000000e-07*exp(make_hipDoubleComplex(0.0,2.0000000000000000e-02)*pow(2.0,(1.0/2.0))*pow(kappa*V_0,(1.0/2.0)))*exp(-2.0000000000000000e-02*pow(2.0,(1.0/2.0))*pow( V_0-kappa*V_0,(1.0/2.0)))*kappa*V_0*( V_0-kappa*V_0)+-8.0000000000000002e-08*exp(-2.0000000000000000e-02*pow(2.0,(1.0/2.0))*pow( V_0-kappa*V_0,(1.0/2.0)))*pow( V_0-kappa*V_0,2.0)+make_hipDoubleComplex(0.0,-1.6000000000000000e-07)*exp(-4.0000000000000001e-02*pow(2.0,(1.0/2.0))*pow( V_0-kappa*V_0,(1.0/2.0)))*pow( V_0-kappa*V_0,(1.0/2.0))*pow(kappa*V_0,(3.0/2.0))+-4.0000000000000001e-08*exp(-4.0000000000000001e-02*pow(2.0,(1.0/2.0))*pow( V_0-kappa*V_0,(1.0/2.0)))*exp(make_hipDoubleComplex(0.0,2.0000000000000000e-02)*pow(2.0,(1.0/2.0))*pow(kappa*V_0,(1.0/2.0)))*pow( V_0-kappa*V_0,2.0)+4.0000000000000001e-08*(kappa*kappa)*(V_0*V_0)+-4.0000000000000001e-08*exp(-4.0000000000000001e-02*pow(2.0,(1.0/2.0))*pow( V_0-kappa*V_0,(1.0/2.0)))*exp(make_hipDoubleComplex(0.0,2.0000000000000000e-02)*pow(2.0,(1.0/2.0))*pow(kappa*V_0,(1.0/2.0)))*(kappa*kappa)*(V_0*V_0)+-8.0000000000000002e-08*exp(-4.0000000000000001e-02*pow(2.0,(1.0/2.0))*pow( V_0-kappa*V_0,(1.0/2.0)))*exp(make_hipDoubleComplex(0.0,2.0000000000000000e-02)*pow(2.0,(1.0/2.0))*pow(kappa*V_0,(1.0/2.0)))*kappa*V_0*( V_0-kappa*V_0)+-8.0000000000000002e-08*exp(-2.0000000000000000e-02*pow(2.0,(1.0/2.0))*pow( V_0-kappa*V_0,(1.0/2.0)))*(kappa*kappa)*(V_0*V_0)+4.0000000000000001e-08*pow( V_0-kappa*V_0,2.0)+8.0000000000000002e-08*exp(make_hipDoubleComplex(0.0,2.0000000000000000e-02)*pow(2.0,(1.0/2.0))*pow(kappa*V_0,(1.0/2.0)))*exp(-2.0000000000000000e-02*pow(2.0,(1.0/2.0))*pow( V_0-kappa*V_0,(1.0/2.0)))*pow( V_0-kappa*V_0,2.0)+-4.0000000000000001e-08*exp(make_hipDoubleComplex(0.0,2.0000000000000000e-02)*pow(2.0,(1.0/2.0))*pow(kappa*V_0,(1.0/2.0)))*(kappa*kappa)*(V_0*V_0)+8.0000000000000002e-08*exp(make_hipDoubleComplex(0.0,2.0000000000000000e-02)*pow(2.0,(1.0/2.0))*pow(kappa*V_0,(1.0/2.0)))*exp(-2.0000000000000000e-02*pow(2.0,(1.0/2.0))*pow( V_0-kappa*V_0,(1.0/2.0)))*(kappa*kappa)*(V_0*V_0)+-1.6000000000000000e-07*exp(-2.0000000000000000e-02*pow(2.0,(1.0/2.0))*pow( V_0-kappa*V_0,(1.0/2.0)))*kappa*V_0*( V_0-kappa*V_0))*( V_0-kappa*V_0)
;
}
